#include "hip/hip_runtime.h"
﻿﻿//--------------------------------------------------------------------------------------------------------------------------------------------------
// Project : Real-time-Particle-Simulation-with-CUDA
// Implement a Particle Simulation with CUDA acceleration with real-time visualization.
// Author: Arsheya Raj
// Date: 4th April 2025
//--------------------------------------------------------------------------------------------------------------------------------------------------
//
//	Develope a system that uses CUDA to accelerate the simulation of particle systems. This enables real-time visualization and
//  analysis of complex fluid dynamics or physics simulations on local hardware.
// 
//--------------------------------------------------------------------------------------------------------------------------------------------------

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <fstream>
#include <sstream>

const int NUM_PARTICLES = 10000;    // Increased number of particles
const int NUM_FRAMES = 1000;        // Increased number of frames
const float DT = 0.01f;
const float GRAVITY = -9.81f;
const float ELASTICITY = 0.9f;

// Waterfall effect parameters
const float MAX_X_SPREAD = 5.0f;  // Max horizontal spread in X direction
const float MAX_Z_SPREAD = 5.0f;  // Max horizontal spread in Z direction
const float INIT_Y = 20.0f;       // Starting Y position (top of the waterfall)

// Particle structure
struct Particle {
    float3 position;
    float3 velocity;
};

// CUDA kernel for updating particle positions
__global__ void updateParticles(Particle* particles, int numParticles, float dt, float gravity, float elasticity) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numParticles) {
        Particle& p = particles[idx];

        // Apply gravity to the velocity
        p.velocity.y += gravity * dt;

        // Update particle position based on velocity
        p.position.x += p.velocity.x * dt;
        p.position.y += p.velocity.y * dt;
        p.position.z += p.velocity.z * dt;

        // Bounce particles off the ground with elasticity
        if (p.position.y < 0) {
            p.position.y = 0;
            p.velocity.y *= -elasticity;
        }
    }
}

// Helper function to simulate particles with CUDA
hipError_t simulateParticlesWithCuda(Particle* particles, int numParticles, float dt, float gravity, float elasticity) {
    Particle* d_particles = nullptr;

    hipError_t cudaStatus = hipMalloc((void**)&d_particles, numParticles * sizeof(Particle));
    if (cudaStatus != hipSuccess) return cudaStatus;

    cudaStatus = hipMemcpy(d_particles, particles, numParticles * sizeof(Particle), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) return cudaStatus;

    // Launch kernel to update particles
    int blockSize = 256;
    int numBlocks = (numParticles + blockSize - 1) / blockSize;
    updateParticles << <numBlocks, blockSize >> > (d_particles, numParticles, dt, gravity, elasticity);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) return cudaStatus;

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) return cudaStatus;

    cudaStatus = hipMemcpy(particles, d_particles, numParticles * sizeof(Particle), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) return cudaStatus;

    hipFree(d_particles);
    return hipSuccess;
}

// Function to append particle data to a CSV file
void appendParticlesToCSV(const Particle* particles, int numParticles, int frame, std::ofstream& file) {
    for (int i = 0; i < numParticles; i++) {
        file << frame << "," << i << ","
            << particles[i].position.x << ","
            << particles[i].position.y << ","
            << particles[i].position.z << "\n";
    }
}

int main() {
    Particle particles[NUM_PARTICLES];
    srand((unsigned int)time(0));

    // Initialize particles with random positions for waterfall effect
    for (int i = 0; i < NUM_PARTICLES; i++) {
        particles[i].position = make_float3(
            (rand() % 2 == 0 ? 1 : -1) * (rand() % (int)(MAX_X_SPREAD * 2)),  // Random X position (spread in range [-MAX_X_SPREAD, MAX_X_SPREAD])
            INIT_Y,  // Y starts at top (higher)
            (rand() % 2 == 0 ? 1 : -1) * (rand() % (int)(MAX_Z_SPREAD * 2))   // Random Z position (spread in range [-MAX_Z_SPREAD, MAX_Z_SPREAD])
        );

        particles[i].velocity = make_float3(
            rand() % 10 - 5,  // Random horizontal velocity
            0.0f,             // Initial Y velocity is 0 (falling due to gravity)
            rand() % 10 - 5   // Random Z velocity
        );
    }

    // Open CSV file to save particle positions
    std::ofstream file("particles.csv");
    if (!file.is_open()) {
        fprintf(stderr, "Failed to open CSV file!\n");
        return 1;
    }
    file << "frame,particle_id,x,y,z\n";

    // Simulate and record the positions of particles over time
    for (int frame = 0; frame < NUM_FRAMES; frame++) {
        hipError_t cudaStatus = simulateParticlesWithCuda(particles, NUM_PARTICLES, DT, GRAVITY, ELASTICITY);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "simulateParticlesWithCuda failed at frame %d!\n", frame);
            return 1;
        }

        // Save particle positions to CSV
        appendParticlesToCSV(particles, NUM_PARTICLES, frame, file);
        printf("Saved frame %d\n", frame);
    }

    // Close the CSV file
    file.close();
    return 0;
}
